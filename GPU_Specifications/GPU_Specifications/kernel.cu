#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "handle_error.h"
#include <stdio.h>
#include <iostream>

__global__ void kernel(void) {
}
int getSPcores(hipDeviceProp_t devProp)
{
	int cores = 0;
	int mp = devProp.multiProcessorCount;
	switch (devProp.major){
	case 2: // Fermi
		if (devProp.minor == 1) cores = mp * 48;
		else cores = mp * 32;
		break;
	case 3: // Kepler
		cores = mp * 192;
		break;
	case 5: // Maxwell
		cores = mp * 128;
		break;
	default:
		printf("Unknown device type\n");
		break;
	}
	return cores;
}
int main(void){
	
	int gpu_count;
	HANDLE_ERROR(hipGetDeviceCount(&gpu_count));
	printf("This computer has %d CUDA capable GPU units\n", gpu_count);
	for (int i = 0; i < gpu_count; i++) {
		hipDeviceProp_t prop;
		int cuda_cores;
		HANDLE_ERROR(hipGetDeviceProperties(&prop, i));
		printf("---------------------------------------------\n");
		printf("GPU unit %d\n", i+1);
		printf("---------------------------------------------\n");
		printf("Name: %s\n", prop.name);
		cuda_cores = getSPcores(prop);
		printf("CUDA Cores %d\n", cuda_cores);
		printf("Total global mem: %d\n", prop.totalGlobalMem);
	}

	system("PAUSE");
	return 0;

}