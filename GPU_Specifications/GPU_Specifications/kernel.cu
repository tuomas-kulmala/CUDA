#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "handle_error.h"
#include <stdio.h>
#include <iostream>

__global__ void kernel(void) {
}

int main(void){
	hipDeviceProp_t prop;
	int count;
	HANDLE_ERROR(hipGetDeviceCount(&count));
	for (int i = 0; i < count; i++) {
		HANDLE_ERROR(hipGetDeviceProperties(&prop, i));
	}
	//kernel << <1, 1 >> >();

	printf("hello world");
	system("PAUSE");
	return 0;

}